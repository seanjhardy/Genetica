#include "hip/hip_runtime.h"
#include <geneticAlgorithm/lifeform.hpp>
#include <simulator/simulator.hpp>
#include <geneticAlgorithm/cellParts/cell.hpp>
#include <geneticAlgorithm/cellParts/cellLink.hpp>
#include "geneticAlgorithm/systems/morphology/geneRegulatoryNetwork.hpp"

// This uses a triangular matrix to store distances between each pair of cells
// This is because the distance between cell i and cell j is the same as the distance between cell j and cell i
// This reduces the amount of memory needed to store the distances
// The formula for the linear index of the distance between cell i and cell j is:
// i * numCells - (i * (i + 1)) / 2 + (j - i - 1)
__global__ void calculateDistances(const StaticGPUVector<Cell*> cells, const GPUVector<Point> points, StaticGPUVector<float> output) {
    size_t idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idx2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx1 >= cells.size() || idx2 >= cells.size() || idx1 >= idx2) return;

    size_t linearIdx = idx1 * cells.size() - (idx1 * (idx1 + 1)) / 2 + (idx2 - idx1 - 1);
    auto cell1 = *(cells + idx1);
    auto cell2 = *(cells + idx2);
    const Point* a = points + cell1->pointIdx;
    const Point* b = points + cell2->pointIdx;
    output[linearIdx] = a->distanceTo(*b);
}

__device__ float getCellDistance(const int cellIdx, const int otherCellIdx,
                                 const size_t numCells, const StaticGPUVector<float> cellDistances) {
    size_t linearIdx = cellIdx * numCells - (cellIdx * (cellIdx + 1)) / 2 + (otherCellIdx - cellIdx - 1);
    return cellDistances[linearIdx];
};


__global__ void updateProductConcentration(GeneRegulatoryNetwork grn,
                                           const StaticGPUVector<Cell*> cells,
                                           const GPUVector<Point> points,
                                           const int simulationStep,
                                           const int birthdate,
                                           const float energy) {
    int productIdx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t cellIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (productIdx >= grn.factors.size() || cellIdx >= cells.size()) return;

    //Head
    auto head = *cells[0];
    auto headPos = points[head.pointIdx].getPos();

    float decayRate = 0.99;
    auto cell = *(cells + cellIdx);
    float* amount = cell->products + productIdx;
    Gene* product = grn.factors + productIdx;
    Point* p1 = points + cell->pointIdx;


    if (cell->frozen) return;

    Gene* factor = grn.factors + productIdx;

    // Update product quantities in cell
    if (factor->factorType == Gene::FactorType::MaternalFactor) {
        float2 factorPos = rotate(factor->extra * 10.0f, head.rotation) + headPos;
        *amount = distanceBetween(factorPos, p1->getPos());
    }
    if (factor->factorType == Gene::FactorType::Time) {
        *amount = product->extra.y * (factor->sign ? 1.0f : -1.0f)
          + product->extra.x * (simulationStep - birthdate)/100000.0;
    }
    if (factor->factorType == Gene::FactorType::Constant) {
        *amount = factor->extra.x;
    }
    if (factor->factorType == Gene::FactorType::Generation) {
        *amount = cell->generation;
    }
    if (factor->factorType == Gene::FactorType::Energy) {
        *amount = energy * max(factor->extra.x, 0.1f);
    }
    /*if (factor->factorType == Gene::FactorType::Crowding) {
        *amount = 0;
        for (const auto& otherCell: lifeForm->cells) {
            if (otherCell->frozen) continue;
            if (&otherCell == &cell) continue;
            float distance = cellDistances.at({cell, otherCell});
            *amount += 1.0f / (1.0f + distance);
        }
    }*/
    //Decay products
    if (factor->factorType == Gene::FactorType::InternalProduct) {
        *amount *= decayRate;
    }
}

__global__ void updateNSquaredProductConcentration(
  GeneRegulatoryNetwork grn, const StaticGPUVector<Cell*> cells, const GPUVector<Point> points) {
    size_t productIdx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t cellIdx = blockIdx.y * blockDim.y + threadIdx.y;
    size_t cellIdx2 = blockIdx.z * blockDim.z + threadIdx.z;

    if (productIdx >= grn.factors.size() || cellIdx >= cells.size() || cellIdx2 >= cells.size()) return;

    float decayRate = 0.99;
    auto cell = *(cells + cellIdx);
    auto otherCell = *(cells + cellIdx2);

    float* amount = cell->products + productIdx;
    float otherAmount = otherCell->products[productIdx];

    if (cell->frozen) return;
    if (otherCell->frozen) return;
    if (cellIdx == cellIdx2) return;

    Point* p1 = points + cell->pointIdx;
    Point* p2 = points + otherCell->pointIdx;
    float2 divisionVector = {0, 0};
    Gene* factor = grn.factors + productIdx;

    float cellDistance = getCellDistance(cellIdx, cellIdx2, cells.size(), grn.cellDistances);

    if (factor->factorType == Gene::FactorType::ExternalProduct) {
        float distanceScale = 1.0f / (1.0f + cellDistance);
        *amount += distanceScale * otherAmount;
    }

    if (factor->factorType == Gene::FactorType::Receptor && cellIdx != 0) {
        float2 normalisedVectorToCell = p1->getPos() - p2->getPos() / cellDistance;

        for(int i = 0; i < grn.factors.size(); i++) {
            if (grn.factors[i].factorType != Gene::FactorType::ExternalProduct) continue;
            float receptorAmount = otherCell->products[i];
            int affinityIndex = (int)(productIdx * grn.factors.size() + i);
            float affinity = grn.factorReceptorAffinities[affinityIndex];
            divisionVector += (*amount) * affinity * receptorAmount * normalisedVectorToCell;
        }
        cell->divisionRotation = std::atan2(divisionVector.y, divisionVector.x);
    }

    *amount *= decayRate;
}

__global__ void updateRegulatoryUnitExpression(GeneRegulatoryNetwork grn,
                                               const StaticGPUVector<Cell*> cells,
                                               const GPUVector<Point> points) {
    size_t cellIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (cellIdx >= cells.size()) return;

    auto cell = *(cells + cellIdx);
    if (cell->frozen) return;

    // For each cell, update it's products based on the regulatory units
    for (int i = 0; i < grn.regulatoryUnits.size(); i++) {
        float* factorLevels = grn.regulatoryUnits[i].calculateActivation(grn.promoters,
                                                         grn.factors,
                                                         cell->products,
                                                         grn.promoterFactorAffinities);
        // Add factor levels back to cell's products
        for (int j = 0; j < grn.factors.size(); j++) {
            cell->products[j] += factorLevels[j];
        }
    }
}

__global__ void updateGeneExpression(GeneRegulatoryNetwork grn,
    StaticGPUVector<Cell*> cellPtrs,
    StaticGPUVector<CellLink*> cellLinkPtrs,
    const GPUVector<Point> points) {
    size_t cellIdx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t effectorIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (cellIdx >= cellPtrs.size() || effectorIdx >= grn.effectors.size()) return;

    auto cell = *(cellPtrs + cellIdx);
    auto effector = grn.effectors + effectorIdx;

    if (cell->frozen) return;

    float expression = 0.0f;
    for (int productIdx = 0; productIdx < cell->products.size(); productIdx++) {
        auto gene = grn.factors + productIdx;
        auto level = cell->products[productIdx];
        if (gene->factorType != Gene::FactorType::InternalProduct) continue;
        int affinityIndex = productIdx * grn.factors.size() + effectorIdx;
        expression += level * grn.factorEffectorAffinities[affinityIndex];
    }

    if (expression == 0) return;

    /*if (effector->effectorType == Effector::EffectorType::Die) {
        if (expression > 0.5 && cellIdx != 0) {
            cell->dead = true;
        }
    }
    if (effector->effectorType == Effector::EffectorType::Divide) {
        if (expression > 0.0001 && !cell->dividing) {
            cell->dividing = true;
        }
    }
    if (effector->effectorType == Effector::EffectorType::Freeze) {
        if (expression > 0.5) {
            cell->frozen = true;
        }
    }
    if (effector->effectorType == Effector::EffectorType::Distance) {*/
        /*for (auto cellLink : cellLinkPtrs) {
            if (cellLink->cellAId == cellIdx || cellLink->cellBId == cellIdx) {
                cellLink->adjustSize(min(expression, 0.0f));
            }
        }*/
    /*}
    if (effector->effectorType == Effector::EffectorType::Radius) {
        float sizeChange = expression;
        int pointIdx = cell->pointIdx;
        Point* pointObj = points + pointIdx;
        atomicAdd(&cell->energyUse, 2.0f * M_PI * pointObj->radius * sizeChange + M_PI * pow(sizeChange, 2));
        if (pointObj->radius + sizeChange < 20) {
            pointObj->radius = max(pointObj->radius + sizeChange, 0.5f);
        }
    }
    if (effector->effectorType == Effector::EffectorType::Red) {
        cell->updateHue(Red, expression * 0.1f);
    }
    if (effector->effectorType == Effector::EffectorType::Green) {
        cell->updateHue(Green, expression * 0.1f);
    }
    if (effector->effectorType == Effector::EffectorType::Blue) {
        cell->updateHue(Blue, expression * 0.1f);
    }*/
}

void updateGRN(LifeForm& lifeForm,
               GPUVector<Cell>& cells,
               GPUVector<CellLink>& cellLinks,
               GPUVector<Point>& points) {
    // Construct static subsets of life form cells and links of the lifeform
    std::vector<Cell*> hostCellPtrs;
    for (int i = 0; i < lifeForm.cells.size(); i++) {
        hostCellPtrs.push_back(&cells[lifeForm.cells[i]]);
    }
    StaticGPUVector cellsPtrs(hostCellPtrs);

    std::vector<CellLink*> hostCellLinkPtrs;
    for (int i = 0; i < lifeForm.links.size(); i++) {
        hostCellLinkPtrs.push_back(&cellLinks[lifeForm.links[i]]);
    }
    StaticGPUVector cellLinkPtrs(hostCellLinkPtrs);


    // Calculate distances between each pair of cells
    lifeForm.grn.cellDistances.destroy();
    lifeForm.grn.cellDistances = StaticGPUVector<float>((cellsPtrs.size() * (cellsPtrs.size() - 1)) / 2);
    dim3 threadsPerBlock(32, 32);
    dim3 numDistanceBlocks((cellsPtrs.size() + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (cellsPtrs.size() + threadsPerBlock.y - 1) / threadsPerBlock.y);
    calculateDistances<<<numDistanceBlocks, threadsPerBlock>>>(cellsPtrs, points, lifeForm.grn.cellDistances);
    hipDeviceSynchronize();

    // Update product concentration
    dim3 numProductBlocks((lifeForm.grn.factors.size() + threadsPerBlock.x - 1) / threadsPerBlock.x,
                          (hostCellPtrs.size() + threadsPerBlock.y - 1) / threadsPerBlock.y);
    updateProductConcentration<<<numProductBlocks, threadsPerBlock>>>(
        lifeForm.grn,
        cellsPtrs,
        points,
        Simulator::get().getStep(),
        lifeForm.birthdate,
        lifeForm.energy);

    // Update product concentration based on n squared interactions
    dim3 threadsPerCellProductBlock(32, 32, 32);
    dim3 numNSquaredProductBlocks((lifeForm.grn.factors.size() + threadsPerCellProductBlock.x - 1) / threadsPerCellProductBlock.x,
                                  (hostCellPtrs.size() + threadsPerCellProductBlock.y - 1) / threadsPerCellProductBlock.y,
                                  (hostCellPtrs.size() + threadsPerCellProductBlock.z - 1) / threadsPerCellProductBlock.z);
    updateNSquaredProductConcentration<<<numNSquaredProductBlocks, threadsPerCellProductBlock>>>(lifeForm.grn, cellsPtrs, points);

    // Update each cell's products based on regulatory expression
    size_t numCellBlocks((hostCellPtrs.size() + threadsPerBlock.x - 1) / threadsPerBlock.x);
    updateRegulatoryUnitExpression<<<numCellBlocks, threadsPerBlock>>>(lifeForm.grn, cellsPtrs, points);

    // Update gene expression based on the products
    /*dim3 numEffectorBlocks((hostCellPtrs.size() + threadsPerBlock.x - 1) / threadsPerBlock.x,
                        (lifeForm.grn.effectors.size() + threadsPerBlock.y - 1) / threadsPerBlock.y);
    updateGeneExpression<<<numEffectorBlocks, threadsPerBlock>>>(lifeForm.grn, cellsPtrs, cellLinkPtrs, points);*/
}