
#include <hip/hip_runtime.h>
// Error checking macro
#define cudaCheckError() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}