#include "hip/hip_runtime.h"
#include "../verlet/point.hpp"
#include "../verlet/constraints.cu"
#include <hip/hip_runtime.h>
#include <>
#include "updatePoints.hpp"

__global__ void updatePointsKernel(Point* points, int numParticles, const sf::FloatRect& bounds, float dt) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < numParticles) {
        Point& point = points[index];
        constrainPosition(point, bounds);
        point.update(dt);
    }
}

void updatePointsOnGPU(std::vector<Point>& points, const sf::FloatRect& bounds, float dt) {
    Point* d_points;
    sf::FloatRect* d_bounds;

    size_t pointsSize = points.size() * sizeof(Point);
    size_t boundsSize = sizeof(float2);

    hipMalloc(&d_points, pointsSize);
    hipMalloc(&d_bounds, boundsSize);

    hipMemcpy(d_points, points.data(), pointsSize, hipMemcpyHostToDevice);
    hipMemcpy(d_bounds, &bounds, boundsSize, hipMemcpyHostToDevice);

    int numPoints = points.size();
    int blockSize = 256; // Number of threads per block
    int numBlocks = (numPoints + blockSize - 1) / blockSize;

    updatePointsKernel<<<numBlocks, blockSize>>>(d_points, numPoints, *d_bounds, dt);

    hipMemcpy(points.data(), d_points, pointsSize, hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_bounds);
}