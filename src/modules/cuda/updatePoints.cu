#include "hip/hip_runtime.h"
#include <modules/physics/point.hpp>
#include "../physics/constraints.cu"
#include "hip/hip_runtime.h"
#include <modules/cuda/updatePoints.hpp>
#include <SFML/Graphics.hpp>

__global__ void updatePointsKernel(GPUVector<Point> points, sf::FloatRect* bounds) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= points.size()) return;

    Point& point = points[index];
    point.update();
    constrainPosition(point, *bounds);
}

// Soft collisions (resistive forces)
__global__ void computeCollisions(GPUVector<Point> points) {
    int a = blockIdx.x * blockDim.x + threadIdx.x;
    int b = blockIdx.y * blockDim.y + threadIdx.y;

    if (a >= points.size() || b >= points.size() || a >= b) return;

    Point& pointA = points[a];
    Point& pointB = points[b];

    if (pointA.entityID == pointB.entityID) return;

    constrainMinDistance(pointA, pointB, pointA.radius + pointB.radius);
}

void updatePoints(GPUVector<Point>& points,
                  CGPUValue<sf::FloatRect>& bounds) {
    int blockSize = 256; // Number of threads per block
    int numBlocks = 0;

    if (points.size() == 0) return;

    // Update the points
    numBlocks = (points.size() + blockSize - 1) / blockSize;
    updatePointsKernel<<<numBlocks, blockSize>>>(points, bounds.deviceData());

    dim3 threadsPerBlock(32, 32);
    dim3 numCollisionBlocks((points.size() + threadsPerBlock.x - 1) / threadsPerBlock.x,
                            (points.size() + threadsPerBlock.y - 1) / threadsPerBlock.y);
    computeCollisions<<<numCollisionBlocks, threadsPerBlock>>>(points);
}

__global__ void movePointKernel(GPUVector<Point> points, int pointIndex, const float2 newPos, int* entityID) {
    Point& point = points[pointIndex];
    point.pos.x = newPos.x;
    point.pos.y = newPos.y;
    *entityID = point.entityID;
}

int movePoint(GPUVector<Point>& points, int pointIndex, const sf::Vector2f& newPos) {
    int* entityID = nullptr;
    cudaLog(hipMalloc(&entityID, sizeof(int)));
    movePointKernel<<<1, 1>>>(points, pointIndex, {newPos.x, newPos.y}, entityID);
    int entityIDHost;
    hipMemcpy(&entityIDHost, entityID, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(entityID);
    return entityIDHost;
}
