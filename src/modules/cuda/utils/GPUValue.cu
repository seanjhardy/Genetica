#include <vector>
#include <hip/hip_runtime.h>
#include <>
#include "GPUValue.hpp"


template<typename T>
GPUValue<T>::GPUValue(T& value) {
    h_data = value;
    hipMemcpy(d_data, &value, sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
GPUValue<T>::~GPUValue() {
    if (d_data) {
        hipFree(d_data);
    }
}

template<typename T>
GPUValue<T>& GPUValue<T>::operator=(T& value) {
    h_data = value;
    hipMemcpy(d_data, &value, sizeof(T), hipMemcpyHostToDevice);
}


template<typename T>
void GPUValue<T>::syncToHost() {
    hipMemcpy(h_data.data(), d_data, sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void GPUValue<T>::syncToDevice() {
    hipMemcpy(d_data, h_data.data(), sizeof(T), hipMemcpyHostToDevice);
}
